#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <time.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/time.h>

#include "execute.h"

/* This variable is used to check if there should be next generation */
__device__ int diff = 0;

void execute(int dimension, int loops, char *input_file, int prints_enabled) {

	int local_diff;
	char *grid, *gpu_grid_1, *gpu_grid_2;

	struct timeval time_1, time_2;
	gettimeofday(&time_1, 0);

	createGrid(&grid, dimension);
	if (input_file != NULL) {
		readGrid(grid, input_file, dimension);
	} else {
		initGrid(grid, dimension);
	}
	
	if (prints_enabled == 1) {
		int dir_stat = mkdir("outputs", S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
		if (dir_stat != 0 && errno != EEXIST) {
		printf("mkdir error %s\n", strerror(errno));
			exit(EXIT_FAILURE);
		}
		printGrid(grid, dimension);
	}

	hipMalloc((void **) &gpu_grid_1, dimension * dimension * sizeof(char));
	hipMalloc((void **) &gpu_grid_2, dimension * dimension * sizeof(char));
	hipMemcpy(gpu_grid_1, grid, dimension * dimension * sizeof(char), hipMemcpyHostToDevice);

	/* Kernel invocation */
	dim3 dimBlock(16, 16);
	dim3 dimGrid;
	dimGrid.x = (dimension + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (dimension + dimBlock.y - 1) / dimBlock.y;

	int generation = 1;
	while (generation <= loops) {
		if (prints_enabled == 1) {
			printf("Generation: %d\n", generation);
		}
		local_diff = 0;
		hipMemcpyToSymbol(HIP_SYMBOL(diff), &local_diff,sizeof(int), 0, hipMemcpyHostToDevice);

		kernel<<<dimGrid, dimBlock>>>(gpu_grid_1, gpu_grid_2, dimension);

		if (hipGetLastError() != hipSuccess) {
			printf("kernel launch failed\n");
		}
		hipDeviceSynchronize();
		
		//todo 8elei optimize auto
		if (prints_enabled == 1) {
			hipMemcpy(grid, gpu_grid_2, dimension * dimension * sizeof(char), hipMemcpyDeviceToHost);
			printGrid(grid, dimension);
		}

		hipMemcpyFromSymbol(&local_diff, HIP_SYMBOL(diff), sizeof(int), 0, hipMemcpyDeviceToHost);

		// printf("local_diff %d\n", local_diff);

		/* If there are no differences between two generations
		 * OR if the next generation is 0 */
		if(local_diff == 0) {
			break;
		}

		char *temp = gpu_grid_1;
		gpu_grid_1 = gpu_grid_2;
		gpu_grid_2 = temp;

		generation++;
	}

	gettimeofday(&time_2, 0);
	double time = (1000000.0 * (time_2.tv_sec - time_1.tv_sec) + time_2.tv_usec - time_1.tv_usec) / 1000000;
	printf("time elapsed: %lf\n", time);

	hipFree(gpu_grid_1);
	hipFree(gpu_grid_2);
	freeGrid(&grid);
}

__global__ void kernel(char *grid_1, char *grid_2, int dimension) {

	/* The variables below are used to iterate the grid */
	int ix = (blockIdx.x * blockDim.x + threadIdx.x) % (dimension * dimension);
	int iy = (blockIdx.y * blockDim.y + threadIdx.y) % (dimension * dimension);
	int idx	= (iy * dimension + ix) % (dimension * dimension);

	int i = idx / dimension;
	int j = idx % dimension;

	int top_offset = ((i + dimension - 1) % dimension) * dimension;
	int bot_offset = ((i + 1) % dimension) * dimension;
	int right_offset = (j + 1) % dimension;
	int left_offset = (j - 1 + dimension) % dimension;

	int top = top_offset + j;
	int top_right = top_offset + right_offset;
	int top_left = top_offset + left_offset;

	int bot = bot_offset + j;
	int bot_right = bot_offset + right_offset;
	int bot_left = bot_offset + left_offset;

	int right = i * dimension + right_offset;
	int left = i * dimension + left_offset;

	int alive_neighbors = 0;
	alive_neighbors += grid_1[top_left];
	alive_neighbors += grid_1[top];
	alive_neighbors += grid_1[top_right];
	alive_neighbors += grid_1[right];
	alive_neighbors += grid_1[bot_right];
	alive_neighbors += grid_1[bot];
	alive_neighbors += grid_1[bot_left];
	alive_neighbors += grid_1[left];

	int status = grid_1[idx];
	// printf("status %d\n", grid_1[idx]);

	if (status == 0) {
		/* If there are exactly 3 neighbors create a new cell */
		if (alive_neighbors == 3) {
			/* CREATE NEW CELL */
			grid_2[idx] = 1;
		}
		/* Leave it empty */
		else {
			grid_2[idx] = 0;
		}
	}
	/* If a cell already lives */
	else {
		/* Determine if the cell lives or dies in next round */
		/* DIE */
		if (alive_neighbors < 2 || alive_neighbors > 3) {
			grid_2[idx] = 0;
		}
		/* LIVE */
		else {
			grid_2[idx] = 1;
		}
	}

	/* We don't care about race conditions, we only check if it is different than 0 */
	if (grid_1[idx] != grid_2[idx]) {
		if(grid_2[idx] != 0){
			diff += 1;
		}
	}
}
