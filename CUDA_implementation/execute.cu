#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <time.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>

#include "execute.h"

void execute(int dimension, int loops, char *input_file, int prints_enabled) {
	int i, j, p;
	int nblocks;
	char **grid, **gpu_grid;
	createGrid(&grid, dimension);
	initGrid(grid, dimension);
	if (input_file != NULL) {
		readGrid(grid, input_file, dimension);
	}

	for(i = 0; i < dimension; i++) {
		for(j = 0; j < dimension; j++) {
			printf("%d ", grid[i][j]);
		}
		printf("\n");
	}
	printf("\nEND OF PRINT\n");
	fflush(stdout);

	//todo
	int dir_stat = mkdir("outputs", S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
	if (dir_stat != 0 && errno != EEXIST) {
	printf("mkdir error %s\n", strerror(errno));
		exit(EXIT_FAILURE);
	}
	hipMalloc(&gpu_grid, dimension * dimension * sizeof(char));
	hipMemcpy(grid, gpu_grid, dimension * dimension * sizeof(char), hipMemcpyHostToDevice);
	kernel<<<128, dimension * dimension / 128>>>(grid, dimension);
	hipMemcpy(gpu_grid, grid, dimension * dimension * sizeof(char), hipMemcpyDeviceToHost);
	
	for(i = 0; i < dimension; i++) {
		for(j = 0; j < dimension; j++) {
			printf("%d ", grid[i][j]);
		}
		printf("\n");
	}
	printf("\nEND OF PRINT\n");
	fflush(stdout);
	// nblocks = ()

}

__global__ void kernel(char **grid, int dimension) {
	printf("EXECUTING KERNEL\n");
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int	iy = blockIdx.y*blockDim.y + threadIdx.y;
	int idx	= iy * dimension + ix;
	printf("KERNEL: %d\n", grid[ix][iy]);
	if(grid[ix][iy] == 1){
		grid[ix][iy] = 2;
	}
}