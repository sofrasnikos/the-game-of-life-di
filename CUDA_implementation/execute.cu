#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <time.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>

#include "execute.h"

void execute(int dimension, int loops, char *input_file, int prints_enabled) {
	int i, j, p;
	int nblocks;
	// char grid[dimension][dimension];
	char *grid, *gpu_grid_1, *gpu_grid_2;
	createGrid(&grid, dimension);
	initGrid(grid, dimension);
	grid[55] = 1;
	grid[45] = 1;
	grid[35] = 1;
	grid[56] = 1;
	grid[54] = 1;
	
	// if (input_file != NULL) {
	// 	readGrid(grid, input_file, dimension);
	// }

	for (i = 0; i < dimension; i++) {
		for (j = 0; j < dimension; j++) {

			printf("%d", grid[i * dimension + j]);
		}
		printf("\n");
	}

	printf("\nEND OF PRINT\n");
	fflush(stdout);

	//todo
	int dir_stat = mkdir("outputs", S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
	if (dir_stat != 0 && errno != EEXIST) {
	printf("mkdir error %s\n", strerror(errno));
		exit(EXIT_FAILURE);
	}

	hipMalloc((void **) &gpu_grid_1, dimension * dimension * sizeof(char));
	hipMalloc((void **) &gpu_grid_2, dimension * dimension * sizeof(char));
	hipMemcpy(gpu_grid_1, grid, dimension * dimension * sizeof(char), hipMemcpyHostToDevice);
	kernel<<<128, (dimension * dimension / 128) + 1>>>(gpu_grid_1, gpu_grid_2, dimension);
	hipMemcpy(grid, gpu_grid_2, dimension * dimension * sizeof(char), hipMemcpyDeviceToHost);
	
	for(i = 0; i < dimension; i++) {
		for(j = 0; j < dimension; j++) {
			printf("%d", grid[i * dimension + j]);
		}
		printf("\n");
	}

	printf("\nEND OF PRINT 2\n");
	fflush(stdout);
	// nblocks = ()
	freeGrid(&grid);
	printf("Exiting...\n");
}

__global__ void kernel(char *grid_1, char *grid_2, int dimension) {

	int ix = (blockIdx.x * blockDim.x + threadIdx.x) % (dimension * dimension);
	int iy = (blockIdx.y * blockDim.y + threadIdx.y) % (dimension * dimension);
	int idx	= (iy * dimension + ix) % (dimension * dimension);

	int i = idx / dimension;
	int j = idx % dimension;

	int top_offset = ((i + dimension - 1) % dimension) * dimension;
	int bot_offset = ((i + 1) % dimension) * dimension;
	int right_offset = (j + 1) % dimension;
	int left_offset = (j - 1 + dimension) % dimension;

	int top = top_offset + j;
	int top_right = top_offset + right_offset;
	int top_left = top_offset + left_offset;

	int bot = bot_offset + j;
	int bot_right = bot_offset + right_offset;
	int bot_left = bot_offset + left_offset;

	int right = i * dimension + right_offset;
	int left = i * dimension + left_offset;

	int alive_neighbors = 0;
	alive_neighbors += grid_1[top_left];
	alive_neighbors += grid_1[top];
	alive_neighbors += grid_1[top_right];
	alive_neighbors += grid_1[right];
	alive_neighbors += grid_1[bot_right];
	alive_neighbors += grid_1[bot];
	alive_neighbors += grid_1[bot_left];
	alive_neighbors += grid_1[left];
	
	// if (idx == 66){
	// 	for (i = 0; i < dimension; i++) {
	// 		for (j = 0; j < dimension; j++) {

	// 			printf("%d", grid_1[i * dimension + j]);
	// 		}
	// 		printf("\n");
	// 	}
	// }
	printf("idx = %d i = %d j = %d\n", idx, i ,j);

	if (i == 5 && j == 5){
		
		printf("alive alive_neighbors: %d\n", alive_neighbors);
		printf("top: %d\n", top);
		printf("top_right: %d\n", top_right);
		printf("top_left: %d\n", top_left);
	 	printf("bot: %d\n", bot);
	 	printf("bot_right: %d\n", bot_right);
	 	printf("bot_left: %d\n", bot_left);
	 	printf("right: %d\n", right);
	 	printf("left: %d\n", left);
	}
	// grid_2[i * dimension + j] = deadOrAlive(alive_neighbors, grid_1[i * dimension + j]);
	int pos = i * dimension + j;
	int status = grid_1[idx];

	// printf("status %d\n", grid_1[idx]);

	if (status == 0) {
		/* If there are exactly 3 neighbors create a new cell */
		if (alive_neighbors == 3) {
			/* CREATE NEW CELL */
			grid_2[idx] = 1;
		}
		/* Leave it empty */
		else {
			grid_2[idx] = 0;
		}
	}
	/* If a cell already lives */
	else {
		/* Determine if the cell lives or dies in next round */
		/* DIE */
		if (alive_neighbors < 2 || alive_neighbors > 3) {
			grid_2[idx] = 0;
		}
		/* LIVE */
		else {
			grid_2[idx] = 1;
		}
	}
}
