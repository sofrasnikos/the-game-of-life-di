#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <time.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>

#include "execute.h"

void execute(int dimension, int loops, char *input_file, int prints_enabled) {
	int i, j, p;
	int nblocks;
	char grid[dimension][dimension];
	char /***grid,*/ *gpu_grid;
	// createGrid(&grid, dimension);
	// initGrid(grid, dimension);
	// if (input_file != NULL) {
	// 	readGrid(grid, input_file, dimension);
	// }

	for (i = 0; i < dimension; i++) {
		for (j = 0; j < dimension; j++) {
			grid[i][j] = 1;
			printf("%d ", grid[i][j]);
		}
		printf("\n");
	}

	printf("\nEND OF PRINT\n");
	fflush(stdout);

	//todo
	int dir_stat = mkdir("outputs", S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
	if (dir_stat != 0 && errno != EEXIST) {
	printf("mkdir error %s\n", strerror(errno));
		exit(EXIT_FAILURE);
	}
	hipMalloc((void **) &gpu_grid, dimension * dimension * sizeof(char));
	hipMemcpy(gpu_grid , grid, dimension * dimension * sizeof(char), hipMemcpyHostToDevice);
	kernel<<<128, (dimension * dimension / 128) + 1>>>(gpu_grid, dimension);
	hipMemcpy(grid, gpu_grid, dimension * dimension * sizeof(char), hipMemcpyDeviceToHost);
	
	for(i = 0; i < dimension; i++) {
		for(j = 0; j < dimension; j++) {
			printf("%d ", grid[i][j]);
		}
		printf("\n");
	}
	printf("\nEND OF PRINT\n");
	fflush(stdout);
	// nblocks = ()

}

__global__ void kernel(char *grid, int dimension) {
	// printf("EXECUTING KERNEL\n");
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int	iy = blockIdx.y*blockDim.y + threadIdx.y;
	int idx	= (iy * dimension + ix) % (dimension * dimension);
	printf("idx = %d\n", idx);
	grid[idx] += 1;
	// grid[0] = 2;
	// printf("KERNEL: "/*%d\n", grid[0][0]*/);
  	// for (idx = iy * dimension + ix; idx < dimension * dimension; idx += blockDim.x * gridDim.x) {
	// if(grid[idx] == 2){
	// 	printf("changed to 3\n");
	// 	grid[idx] = 3;
	// }
	// if(grid[idx] == 1){
	// 	printf("changed to 2\n");
	// 	grid[idx] = 2;
	// }
	// }
}